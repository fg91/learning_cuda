#include "hip/hip_runtime.h"
 //****************************************************************************

// For a color image that has multiple channels, we suggest separating
// the different color channels so that each color is stored contiguously
// instead of being interleaved. This will simplify your code.

// That is instead of RGBARGBARGBARGBA... we suggest transforming to three
// arrays (as in the previous homework we ignore the alpha channel again):
//  1) RRRRRRRR...
//  2) GGGGGGGG...
//  3) BBBBBBBB...
//
// The original layout is known an Array of Structures (AoS) whereas the
// format we are converting to is known as a Structure of Arrays (SoA).

// As a warm-up, we will ask you to write the kernel that performs this
// separation. You should then write the "meat" of the assignment,
// which is the kernel that performs the actual blur. We provide code that
// re-combines your blurred results for each color channel.

//****************************************************************************

// A good starting place is to map each thread to a pixel as you have before.
// Then every thread can perform steps 2 and 3 in the diagram above
// completely independently of one another.

// Note that the array of weights is square, so its height is the same as its width.
// We refer to the array of weights as a filter, and we refer to its width with the
// variable filterWidth.

//****************************************************************************

// Your homework submission will be evaluated based on correctness and speed.
// We test each pixel against a reference solution. If any pixel differs by
// more than some small threshold value, the system will tell you that your
// solution is incorrect, and it will let you try again.

// Once you have gotten that working correctly, then you can think about using
// shared memory and having the threads cooperate to achieve better performance.

//****************************************************************************

// Also note that we've supplied a helpful debugging function called checkCudaErrors.
// You should wrap your allocation and copying statements like we've done in the
// code we're supplying you. Here is an example of the unsafe way to allocate
// memory on the GPU:
//
// hipMalloc(&d_red, sizeof(unsigned char) * numRows * numCols);
//
// Here is an example of the safe way to do the same thing:
//
// checkCudaErrors(hipMalloc(&d_red, sizeof(unsigned char) * numRows * numCols));
//
// Writing code the safe way requires slightly more typing, but is very helpful for
// catching mistakes. If you write code the unsafe way and you make a mistake, then
// any subsequent kernels won't compute anything, and it will be hard to figure out
// why. Writing code the safe way will inform you as soon as you make a mistake.

// Finally, remember to free the memory you allocate at the end of the function.

//****************************************************************************

#include <>
#include <hip/device_functions.h>
#include <stdio.h>
#include "./utils.h"

#define SHARED
#define FILTERWIDTH 9
#define NTHREADS 32

#ifdef SHARED
__global__
void gaussian_blur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth) {
  // NOTE: Be sure to compute any intermediate results in floating point
  // before storing the final result as unsigned char.

  // NOTE: If a thread's absolute position 2D position is within the image, but some of
  // its neighbors are outside the image, then you will need to be extra careful. Instead
  // of trying to read such a neighbor value from GPU memory (which won't work because
  // the value is out of bounds), you should explicitly clamp the neighbor values you read
  // to be within the bounds of the image. If this is not clear to you, then please refer
  // to sequential reference solution for the exact clamping semantics you should follow.

  // --- copy filter to shared memory --- //
  __shared__ float shared_filter[FILTERWIDTH * FILTERWIDTH];
  if (threadIdx.x < FILTERWIDTH && threadIdx.y < FILTERWIDTH) {
    int idx = threadIdx.x * FILTERWIDTH + threadIdx.y;
    shared_filter[idx] = filter[idx];
  }
  __syncthreads();
  
  const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                       blockIdx.y * blockDim.y + threadIdx.y);
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows) {
    return;
  }
  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
  
  float new_pixel_value = 0.f;
  for (int filter_r = -filterWidth/2; filter_r <= filterWidth/2; ++filter_r) {
    for (int filter_c = -filterWidth/2; filter_c <= filterWidth/2; ++filter_c) {
      int image_r = min(max(thread_2D_pos.y + filter_r, 0), numRows - 1);
      int image_c = min(max(thread_2D_pos.x + filter_c, 0), numCols - 1);
      float pixel_value  = static_cast<float>(inputChannel[image_r * numCols + image_c]);
      float filter_value = shared_filter[(filter_r + filterWidth/2) * filterWidth + filter_c + filterWidth/2];
      new_pixel_value += pixel_value * filter_value;
    }
  }
  outputChannel[thread_1D_pos] = static_cast<unsigned char>(new_pixel_value);
}

#else
__global__
void gaussian_blur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth) {
  // NOTE: Be sure to compute any intermediate results in floating point
  // before storing the final result as unsigned char.

  // NOTE: If a thread's absolute position 2D position is within the image, but some of
  // its neighbors are outside the image, then you will need to be extra careful. Instead
  // of trying to read such a neighbor value from GPU memory (which won't work because
  // the value is out of bounds), you should explicitly clamp the neighbor values you read
  // to be within the bounds of the image. If this is not clear to you, then please refer
  // to sequential reference solution for the exact clamping semantics you should follow.

  const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                       blockIdx.y * blockDim.y + threadIdx.y);
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows) {
    return;
  }
  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
  
  float new_pixel_value = 0.f;
  for (int filter_r = -filterWidth/2; filter_r <= filterWidth/2; ++filter_r) {
    for (int filter_c = -filterWidth/2; filter_c <= filterWidth/2; ++filter_c) {
      int image_r = min(max(thread_2D_pos.y + filter_r, 0), numRows - 1);
      int image_c = min(max(thread_2D_pos.x + filter_c, 0), numCols - 1);
      float pixel_value  = static_cast<float>(inputChannel[image_r * numCols + image_c]);
      float filter_value = filter[(filter_r + filterWidth/2) * filterWidth + filter_c + filterWidth/2];
      new_pixel_value += pixel_value * filter_value;
    }
  }
  outputChannel[thread_1D_pos] = static_cast<unsigned char>(new_pixel_value);
}
#endif  // SHARED

// This kernel takes in an image represented as a uchar4 and splits
// it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel) {
  const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                       blockIdx.y * blockDim.y + threadIdx.y);

  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows) {
    return;
  }

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  uchar4 pixel = inputImageRGBA[thread_1D_pos];

  redChannel[thread_1D_pos]   = pixel.x;
  greenChannel[thread_1D_pos] = pixel.y;
  blueChannel[thread_1D_pos]  = pixel.z;
}

// This kernel takes in three color channels and recombines them
// into one image.  The alpha channel is set to 255 to represent
// that this image has no transparency.
__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols) {
  const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                       blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  // make sure we don't try and access memory outside the image
  // by having any threads mapped there return early
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows) {
    return;
  }
  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  // Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage,
                                const size_t numColsImage,
                                const float* const h_filter,
                                const size_t filterWidth) {
  // allocate memory for the three different channels
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  // filter
  size_t sizeInBytes = sizeof(float) * filterWidth * filterWidth;
  checkCudaErrors(hipMalloc(&d_filter, sizeInBytes));
  
  // Copy the filter on the host (h_filter) to the
  // memory we just allocated on the GPU.
  checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeInBytes, hipMemcpyHostToDevice));
}

void gaussian_blur(const uchar4 * const h_inputImageRGBA,
                   uchar4 * const d_inputImageRGBA,
                   uchar4* const d_outputImageRGBA,
                   const size_t numRows, const size_t numCols,
                   unsigned char *d_redBlurred,
                   unsigned char *d_greenBlurred,
                   unsigned char *d_blueBlurred,
                   const int filterWidth) {
  // Set block size
  const dim3 blockSize(NTHREADS, NTHREADS, 1);

  // Compute correct grid size (i.e., number of blocks per kernel launch)
  const dim3 gridSize(ceil(static_cast<float>(numCols)/NTHREADS),
                      ceil(static_cast<float>(numRows)/NTHREADS), 1);

  // Launch a kernel for separating the RGBA image into different color channels
  separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA, numRows, numCols,
                                            d_red, d_green, d_blue);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // Call the convolution kernel here 3 times, once for each color channel.

  gaussian_blur<<<gridSize, blockSize>>>(d_red, d_redBlurred,
                                         numRows, numCols,
                                         d_filter, filterWidth);

  gaussian_blur<<<gridSize, blockSize>>>(d_green, d_greenBlurred,
                                         numRows, numCols,
                                         d_filter, filterWidth);

  gaussian_blur<<<gridSize, blockSize>>>(d_blue, d_blueBlurred,
                                         numRows, numCols,
                                         d_filter, filterWidth);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // Recombine your results
  recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                             d_greenBlurred,
                                             d_blueBlurred,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  }


// Free all the memory that we allocated
void cleanupCu() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
  checkCudaErrors(hipFree(d_filter));
}
